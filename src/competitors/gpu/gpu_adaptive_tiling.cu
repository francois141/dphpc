#include "hip/hip_runtime.h"
#include "gpu_adaptive_tiling.hpp"

#include <hip/hip_runtime.h>
#include <algorithm>

const int TILE_SIZE = 2;
const int PANEL_SIZE = 3;
const int THRESHOLD = 2;
const int WRAP_SIZE = 2;

/*
This function reorders the columns of the sparse matrix S (CSR) such that all columns with a density
above threshold are at the beginning and all columns with a low denisty are at the end of the panel
*/
__global__ void reorder_csr_row_panel(int* rows, int* cols, float* vals, int* reordered_cols, float* reordered_vals, int* panel_ptr, int* tile_row_ptr, int num_rows, int num_cols){
	int* col_count = (int*)malloc(num_cols * sizeof(int));
	int* col_to_tile_id = (int*)malloc(num_cols * sizeof(int));
	for (int i = 0; i < num_cols; i++){
		col_count[i] = 0;
		col_to_tile_id[i] = 0;
	}


	int start_row = PANEL_SIZE * threadIdx.x;
	int end_row = min(start_row + PANEL_SIZE, num_rows);

	// count the number of non-zero element in current row_panel
	int num_heavy_cols = 0;
	for (int row = start_row; row < end_row; row++){
		int sparse_index = rows[row];
		while (sparse_index < rows[row+1]){
			col_count[cols[sparse_index]]++;

			// store number of heavy columns
			if (col_count[cols[sparse_index]] == THRESHOLD)
				num_heavy_cols++;

			sparse_index++;
		}
	}

	// each heavy tile has TILE_SIZE columns and there is one additional tile for the sparse columns
	int num_tiles = (num_heavy_cols + TILE_SIZE - 1) / TILE_SIZE + 1;

	// run over columns and save the tile_id for each col
	int ctr = 0;
	for (int i = 0; i < num_cols; i++){
		if (col_count[i] >= THRESHOLD){
			col_to_tile_id[i] = ctr / TILE_SIZE;
			ctr++;
		} else {
			col_to_tile_id[i] = num_tiles - 1; // sparse column
		}
	}

	// reoder each row, heavy columns at the front and sparse columns at the back
	for (int row = start_row; row < end_row; row++){
		int sparse_index = rows[row];
		int heavy_ptr = rows[row];
		int sparse_ptr = rows[row+1]-1; // we fill sparse columns from the back
		while (sparse_index < rows[row+1]){
			if (col_count[cols[sparse_index]] >= THRESHOLD){
				reordered_cols[heavy_ptr] = cols[sparse_index];
				reordered_vals[heavy_ptr] = vals[sparse_index];
				heavy_ptr++;
			} else {
				reordered_cols[sparse_ptr] = cols[sparse_index];
				reordered_vals[sparse_ptr] = vals[sparse_index];
				sparse_ptr--;
			}
			sparse_index++;
		}
	}

	// store the number of tiles for each row_pannel and have the the thread with ID 0
	// calculat the prefix sum -> there is an efficient prefix sum implementation for GPUs
	// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
	panel_ptr[threadIdx.x+1] = num_tiles;
	__syncthreads();
	if (threadIdx.x == 0){
		int num_panels = (num_rows + PANEL_SIZE - 1) / PANEL_SIZE + 1;
		for (int i = 1; i < num_panels; i++)
			panel_ptr[i] += panel_ptr[i-1];
	}
	__syncthreads();

	// fill tile_row_ptr for this tile, requires that the prefix sums in panel_ptr are calculated
	// + 1 because we have a dummy first entry set to 0
	int ptr = panel_ptr[threadIdx.x] * PANEL_SIZE + 1;

	// within each row, calculate and store start and end pointer of each tile
	for (int row = start_row; row < end_row; row++){
		int sparse_index = rows[row];
		int tile_at_sparse_index = col_to_tile_id[reordered_cols[sparse_index]];
		for (int tile_id = 0; tile_id < num_tiles; tile_id++){
			
			// move sparse_index to first element of next tile if current tile has elements but we need to stay in current line
			while (sparse_index < rows[row+1] && tile_at_sparse_index == tile_id){
				tile_at_sparse_index = col_to_tile_id[reordered_cols[++sparse_index]];
			}

			// exclusive endpointer for current tile_id
			tile_row_ptr[ptr++] = sparse_index;
		}
	}

	free(col_count);
	free(col_to_tile_id);
}

// perform SDDMM, compute P = (A*B^T) dot S (where dot is the term by term product)
// A is MxK, B is NxK, S and P are MxN sparse
__global__ void gpu_tiled_csr_dense_kernel(float* A, float* B, float* reordered_S, float* P, int* reordered_cols, int* rows, int* panel_ptr, int* tile_row_ptr, int M, int K, int N) {
	int row_panel_id = blockIdx.x;
	int row_offset = threadIdx.x / WRAP_SIZE;
	int slice_base = blockIdx.y * WRAP_SIZE;
	int slice_offset = threadIdx.x % WRAP_SIZE;

	int num_panels = panel_ptr[row_panel_id + 1] - panel_ptr[row_panel_id];

	// don't process the last tile which is sparse and will be handled by different kernel
	for (int tile_id = 0; tile_id < num_panels - 1; tile_id++){

		for (int i = row_offset; i < PANEL_SIZE; i += (blockDim.x + WRAP_SIZE - 1)/WRAP_SIZE){
			int ptr = panel_ptr[row_panel_id] * PANEL_SIZE + i * num_panels + tile_id;

			// iterate over all non zero elements of this row in the given tile
			int low = tile_row_ptr[ptr];
			int high = tile_row_ptr[ptr+1];

			// we slice the K dimension among the thread blocks of the grid.y dimension
			// a thread block always handles WRAP_SIZE elements at a time before moving to the next WRAP_SIZE elements
			for (int k = slice_base + slice_offset; k < K; k += gridDim.y * WRAP_SIZE){
				int abs_row_idx = row_panel_id * PANEL_SIZE + i;
				float element_a = A[abs_row_idx * K + k];
				for (int j = low; j < high; j++){
					// B is transposed
					float val = element_a * B[reordered_cols[j] * K + k];

					// reduce all WRAP elements of the inner product
					for (int l = WRAP_SIZE/2; l >= 1; l /= 2){
						val += __shfl_down(val, k);
					}

					// first thread of each wrap to scale value and update global memory
					// use atomic Add because multiple thread blocks can read and write this value
					if (slice_offset == 0)
						atomicAdd(P + j, val * reordered_S[j]);
				}
			}
		}
	}
}

__global__ void gpu_tiled_csr_sparse_kernel(float* A, float* B, float* reordered_S, float* P, int* reordered_cols, int* rows, int* panel_ptr, int* tile_row_ptr, int M, int K, int N) {
	int row_panel_id = blockIdx.x;
	int row_offset = threadIdx.x / WRAP_SIZE;
	int slice_base = blockIdx.y * WRAP_SIZE;
	int slice_offset = threadIdx.x % WRAP_SIZE;

	// calculate number of tiles before this row panel
	int num_panels = panel_ptr[row_panel_id + 1] - panel_ptr[row_panel_id];

	for (int i = row_offset; i < PANEL_SIZE; i += (blockDim.x + WRAP_SIZE - 1)/WRAP_SIZE){
		// the sparse tile is always the last tile of the given row
		int ptr = panel_ptr[row_panel_id] * PANEL_SIZE + (i+1) * num_panels - 1;
		int low = tile_row_ptr[ptr];
		int high = tile_row_ptr[ptr+1];

		for (int k = slice_base + slice_offset; k < K; k += gridDim.y * WRAP_SIZE){
			int abs_row_idx = row_panel_id * PANEL_SIZE + i;
			float element_a = A[abs_row_idx * K + k];
			for (int j = low; j < high; j++){
				// B is transposed
				float val = element_a * B[reordered_cols[j] * K + k];

				// reduce all WRAP elements of the inner product
				for (int l = WRAP_SIZE/2; l >= 1; l /= 2){
					val += __shfl_down(val, k);
				}

				// first thread of each wrap to scale value and update global memory
				// use atomic Add because multiple thread blocks can read and write this value
				if (slice_offset == 0)
					atomicAdd(P + j, val * reodered_S[j]);
			}
		}
	}
}

template <typename T>
void gpu_adaptive_tiling_csr_wrapper(T* A_gpu, T* B_gpu, T* reordered_S_gpu, T* P_gpu, int* reordered_cols_gpu, int* rows_gpu, int* panel_ptr_gpu, int* tile_row_ptr_gpu, int M, int K, int N) {
	
	dim3 thread_blocks(2,2);
	int num_threads_per_block = 8;

	// Perform SDDMM on the GPU
	gpu_tiled_csr_dense_kernel<<<thread_blocks, num_threads_per_block>>>(A_gpu, B_gpu, reordered_S_gpu, P_gpu, reordered_cols_gpu, rows_gpu, panel_ptr_gpu, tile_row_ptr_gpu, M, K, N);
	gpu_tiled_csr_sparse_kernel<<<thread_blocks, num_threads_per_block>>>(A_gpu, B_gpu, reordered_S_gpu, P_gpu, reordered_cols_gpu, rows_gpu, panel_ptr_gpu, tile_row_ptr_gpu, M, K, N);
}

template <typename T>
void gpu_reorder_csr_row_panel_wrapper(int* rows, int* cols, T* vals, int* reordered_cols, T* reordered_vals, int* panel_ptr, int* tile_row_ptr, int num_rows, int num_cols){
	int num_threads = (num_rows + PANEL_SIZE - 1) / PANEL_SIZE;
	reorder_csr_row_panel<<<1, num_threads>>>(rows, cols, vals, reordered_cols, reordered_vals, panel_ptr, tile_row_ptr, num_rows, num_cols);
}

/* Workaround because the wrappers need to be inside the CUDA file (Would normally write templated functions inside the header file!) */
template void gpu_adaptive_tiling_csr_wrapper<float>(float* A_gpu, float* B_gpu, float* S_gpu, float* P_gpu, int* cols_gpu, int* rows_gpu, int M, int K, int N);

template void gpu_reorder_csr_row_panel_wrapper<float>(int* rows, int* cols, float* vals, int* reordered_cols, float* reordered_vals, int* panel_ptr, int* tile_row_ptr, int num_rows, int num_cols);
