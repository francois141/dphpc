#include "hip/hip_runtime.h"
#include "gpu_adaptive_tiling.hpp"

#include <hip/hip_runtime.h>
#include <algorithm>

const int TILE_SIZE = 256;
const int PANEL_SIZE = 3;
const int THRESHOLD = 2;

/*
This function reorders the columns of the sparse matrix S (CSR) such that all columns with a density
above threshold are at the beginning and all columns with a low denisty are at the end of the panel
*/
__global__ void reorder_csr_row_panel(int* rows, int* cols, float* vals, int* reordered_cols, float* reordered_vals, int* panel_ptr, int num_rows, int num_cols){
	int* col_count = (int*)malloc(num_cols * sizeof(int));
	int* col_to_tile_id = (int*)malloc(num_cols * sizeof(int));
	for (int i = 0; i < num_cols; i++){
		col_count[i] = 0;
		col_to_tile_id[i] = 0;
	}

	int start_row = PANEL_SIZE * threadIdx.x;
	int end_row = min(start_row + PANEL_SIZE, num_rows);

	// count the number of non-zero element in current row_panel
	int num_heavy_cols = 0;
	for (int row = start_row; row < end_row; row++){
		int sparse_index = rows[row];
		while (sparse_index < rows[row+1]){
			col_count[cols[sparse_index]]++;

			// store number of heavy columns
			if (col_count[cols[sparse_index]] == THRESHOLD)
				num_heavy_cols++;

			sparse_index++;
		}
	}

	// each heavy tile has TILE_SIZE columns and there is one additional tile for the sparse columns
	int num_tiles = (num_heavy_cols + TILE_SIZE - 1) / TILE_SIZE + 1;
	panel_ptr[threadIdx.x+1] = num_tiles;

	// run over columns and save the tile_id for each col
	int ctr = 0;
	for (int i = 0; i < num_cols; i++){
		if (col_count[i] >= THRESHOLD){
			col_to_tile_id[i] = ctr / TILE_SIZE;
			ctr++;
		} else {
			col_to_tile_id[i] = num_tiles - 1; // sparse column
		}
	}

	// reoder each row, heavy columns at the front and sparse columns at the back
	for (int row = start_row; row < end_row; row++){
		int sparse_index = rows[row];
		int heavy_ptr = rows[row];
		int sparse_ptr = rows[row+1]-1; // we fill sparse columns from the back
		while (sparse_index < rows[row+1]){
			if (col_count[cols[sparse_index]] >= THRESHOLD){
				reordered_cols[heavy_ptr] = cols[sparse_index];
				reordered_vals[heavy_ptr] = vals[sparse_index];
				heavy_ptr++;
			} else {
				reordered_cols[sparse_index] = cols[sparse_index];
				reordered_vals[sparse_index] = vals[sparse_index];
				sparse_ptr--;
			}
			sparse_index++;
		}
	}
}

// perform SDDMM, compute P = (A*B^T) dot S (where dot is the term by term product)
// A is MxK, B is NxK, S and P are MxN sparse
__global__ void gpu_adaptive_tiling_csr_wrapper(float* A, float* B, float* S, float* P, int* cols, int* rows, int M, int K, int N) {
	int nb_running = gridDim.x * blockDim.x;
}

template <typename T>
void gpu_adaptive_tiling_csr_wrapper(T* A_gpu, T* B_gpu, T* S_gpu, T* P_gpu, int* cols_gpu, int* rows_gpu, int M, int K, int N) {
	
	// Perform SDDMM on the GPU
	// gpu_tiled_csr_kernel<<<32, 512>>>(A_gpu, B_gpu, S_gpu, P_gpu, cols_gpu, rows_gpu, M, K, N);
}

template <typename T>
void gpu_reorder_csr_row_panel_wrapper(int* rows, int* cols, T* vals, int* reordered_cols, T* reordered_vals, int* panel_ptr, int num_rows, int num_cols){
	int num_threads = (num_rows + PANEL_SIZE - 1) / PANEL_SIZE;
	reorder_csr_row_panel<<<1, num_threads>>>(rows, cols, vals, reordered_cols, reordered_vals, panel_ptr, num_rows, num_cols);
}

/* Workaround because the wrappers need to be inside the CUDA file (Would normally write templated functions inside the header file!) */
template void gpu_adaptive_tiling_csr_wrapper<float>(float* A_gpu, float* B_gpu, float* S_gpu, float* P_gpu, int* cols_gpu, int* rows_gpu, int M, int K, int N);

template void gpu_reorder_csr_row_panel_wrapper<float>(int* rows, int* cols, float* vals, int* reordered_cols, float* reordered_vals, int* panel_ptr, int num_rows, int num_cols);
