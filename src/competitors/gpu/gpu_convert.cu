#include "hip/hip_runtime.h"
#include "gpu_convert.hpp"

#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>

constexpr int block_size = 32;

__global__ void gpu_convert_kernel(int* rows, int* rows_coo, int M) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= M) return;
	int start = rows[row];
	int end = rows[row + 1];
	for (int i = start; i < end; i++) {
		rows_coo[i] = row;
	}
}

// perform SDDMM, compute P = (A*B^T) dot S (where dot is the term by term product)
// A is MxK, B is NxK, S and P are MxN sparse
__global__ void gpu_basic_coo_kernel_2(float* A, float* B, float* S, float* P, int* cols, int* rows, int M, int K, int N, int sparse_size) {
	int nb_running = gridDim.x * blockDim.x;
	int min_per_instance = sparse_size / nb_running;
	int leftovers = sparse_size % nb_running;

	// We have to compute sparse_size, each instance will compute a contiguous part of it
	// such that each entrie is computed once and they are evenly distributed
	int instance_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int range_start = min_per_instance * instance_idx + min(instance_idx, leftovers);
	int range_end = min_per_instance * (instance_idx + 1) + min(instance_idx + 1, leftovers);

	// perform the SDDMM algorithm on the range [range_start, range_end[
	for (int entry = range_start; entry < range_end; entry++) {
		int row = rows[entry];
		int col = cols[entry];

		float result = 0.f;
		// matrix multiplication
		for (int i = 0; i < K; i++) {
			// B is transposed
			result += A[row * K + i] * B[col * K + i];
		}
		result *= S[entry];
		P[entry] = result;
	}
}

namespace Competitors {

	void GPUConvert::run_csr(Dense<T>& A, Dense<T>& B, CSR<T>& S, CSR<T>& P) {
		// A is MxK, B is NxK, S and P are MxN sparse
		int M = A.getRows();
		int K = A.getCols();
		int N = B.getRows();

		size_t sparse_size = S.getValues().size();

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);  // Assumes device 0, change if using multiple GPUs

		int num_sm = prop.multiProcessorCount;
		int max_threads_per_sm = prop.maxThreadsPerMultiProcessor;
		int max_thread_blocks_per_sm = prop.maxBlocksPerMultiProcessor;
		int max_threads_per_block = prop.maxThreadsPerBlock;

		// Use maximum number of threads per streaming multiprocessor
		int threads_per_block = std::min(max_threads_per_block, (max_threads_per_sm + max_thread_blocks_per_sm - 1) / max_thread_blocks_per_sm);

		// calculate number of thread blocks by using all available streaming multiprocessors
		int num_thread_blocks = (max_threads_per_sm * num_sm + threads_per_block - 1) / threads_per_block;

		// Convert to COO
		gpu_convert_kernel <<< num_thread_blocks, threads_per_block >>> (rows_gpu, rows_coo_gpu, M);
		// Perform SDDMM on the GPU
		gpu_basic_coo_kernel_2 <<< num_thread_blocks, threads_per_block >>> (A_gpu, B_gpu, S_gpu, P_gpu, cols_gpu, rows_coo_gpu, M, K, N, sparse_size);
		// No need to convert back to CSR, just reuse S

		hipDeviceSynchronize();
	}
}
