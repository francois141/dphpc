#include "hip/hip_runtime.h"
#include "gpu_cuSPARSE.hpp"


__global__ void gpu_cuSPARSE_scale_kernel(float* S, float* P, int S_nnz) {
   	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (; i < S_nnz; i += stride)
		P[i] = S[i] * P[i];

}

template <typename T>
void gpu_cuSPARSE_scale_wrapper(T* S, T* P, int S_nnz) {

	int threads_per_block = 1024;
	int thread_blocks = (S_nnz + threads_per_block - 1) / threads_per_block;

	// Perform SDDMM on the GPU
	gpu_cuSPARSE_scale_kernel<<<thread_blocks, threads_per_block>>>(S, P, S_nnz);
}

/* Workaround because the wrappers need to be inside the CUDA file (Would normally write templated functions inside the header file!) */
template void gpu_cuSPARSE_scale_wrapper<float>(float* S, float* P, int S_nnz);
