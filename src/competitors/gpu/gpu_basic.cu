#include "hip/hip_runtime.h"
#include "gpu_basic.hpp"

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void gpu_basic_csr_kernel(float* A, float* B, float* S, float* P, int* cols, int* rows, int M, int K, int N, int sparse_size, int row_size) {
    int nb_running = gridDim.x * blockDim.x;
    int min_per_instance = (row_size-1) / nb_running;
    int leftovers = row_size % nb_running;

    int instance_idx = blockIdx.x * blockDim.x + threadIdx.x;

    int range_start = min_per_instance * instance_idx + min(instance_idx, leftovers);
    int range_end = min_per_instance * (instance_idx + 1) + min(instance_idx + 1, leftovers);

    for (int row_idx = range_start; row_idx < range_end; row_idx++) {
        int idx = rows[row_idx];

        int row = row_idx;
        while(idx < rows[row_idx+1]) {
            int col = cols[idx];

            float result = 0.f;
            for (int i = 0; i < K; i++) {
                result += A[row * K + i] * B[col * K + i];
            }
            result *= S[idx];
            P[idx] = result;
            idx++;
        }
    }
}

// perform SDDMM, compute P = (A*B^T) dot S (where dot is the term by term product)
// A is MxK, B is NxK, S and P are MxN sparse
__global__ void gpu_basic_coo_kernel(float* A, float* B, float* S, float* P, int* cols, int* rows, int M, int K, int N, int sparse_size) {
	int nb_running = gridDim.x * blockDim.x;
	int min_per_instance = sparse_size / nb_running;
	int leftovers = sparse_size % nb_running;

	// We have to compute sparse_size, each instance will compute a contiguous part of it
	// such that each entrie is computed once and they are evenly distributed
	int instance_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int range_start = min_per_instance * instance_idx + min(instance_idx, leftovers);
	int range_end = min_per_instance * (instance_idx + 1) + min(instance_idx + 1, leftovers);

	// perform the SDDMM algorithm on the range [range_start, range_end[
	for (int entry = range_start; entry < range_end; entry++) {
		int row = rows[entry];
		int col = cols[entry];

		float result = 0.f;
		// matrix multiplication
		for (int i = 0; i < K; i++) {
			// B is transposed
			result += A[row * K + i] * B[col * K + i];
		}
		result *= S[entry];
		P[entry] = result;
	}
}

template <typename T>
void gpu_basic_coo_wrapper(T* A_gpu, T* B_gpu, T* S_gpu, T* P_gpu, int* cols_gpu, int* rows_gpu, int M, int K, int N, int sparse_size) {
	hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // Assumes device 0, change if using multiple GPUs

	int num_sm = prop.multiProcessorCount;
	int max_threads_per_sm = prop.maxThreadsPerMultiProcessor;
	int max_thread_blocks_per_sm = prop.maxBlocksPerMultiProcessor;
	int max_threads_per_block = prop.maxThreadsPerBlock;

	// Use maximum number of threads per streaming multiprocessor
	int threads_per_block = std::min(max_threads_per_block, (max_thread_blocks_per_sm + max_thread_blocks_per_sm - 1) / max_thread_blocks_per_sm);

	// calculate number of thread blocks by using all available streaming multiprocessors
	int num_thread_blocks = (max_threads_per_sm * num_sm + threads_per_block - 1) / threads_per_block;

	std::cout << "num thread blocks " << num_thread_blocks << std::endl;
	std::cout << "num threads per block " << threads_per_block << std::endl;
	// Perform SDDMM on the GPU
	gpu_basic_coo_kernel<<<num_thread_blocks, threads_per_block>>>(A_gpu, B_gpu, S_gpu, P_gpu, cols_gpu, rows_gpu, M, K, N, sparse_size);
}

template <typename T>
void gpu_basic_csr_wrapper(T* A_gpu, T* B_gpu, T* S_gpu, T* P_gpu, int* cols_gpu, int* rows_gpu, int M, int K, int N, int sparse_size, int row_size) {

    // Perform SDDMM on the GPU
    gpu_basic_csr_kernel<<<32, 32>>>(A_gpu, B_gpu, S_gpu, P_gpu, cols_gpu, rows_gpu, M, K, N, sparse_size, row_size);
}

/* Workaround because the wrappers need to be inside the CUDA file (Would normally write templated functions inside the header file!) */
template void gpu_basic_coo_wrapper<float>(float* A_gpu, float* B_gpu, float* S_gpu, float* P_gpu, int* cols_gpu, int* rows_gpu, int M, int K, int N, int sparse_size);
template void gpu_basic_csr_wrapper<float>(float* A_gpu, float* B_gpu, float* S_gpu, float* P_gpu, int* cols_gpu, int* rows_gpu, int M, int K, int N, int sparse_size, int row_size);
